
/* Copyright (c) . All rights reserved. */

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixAdd(const int *data_a, int lda, int k , int *data_o) {

int tid = threadIdx.x;
int bid = blockIdx.x;
int bdm = blockDim.x;

const int *data_a_ptr = data_a + bid*bdm +tid;

int *data_o_ptr = data_o + bid * bdm +tid;

for (int i = 0; i<k; i++) {
  int tmp = *data_a_ptr;
  data_a_ptr += lda;
  *data_o_ptr = tmp +1;
  data_o_ptr += lda;
  }
}

int main(void) {
  hipError_t err = hipSuccess;

  int numElements = 32;
  size_t size = numElements * sizeof(int);

  int *h_data1_in  = (int *)malloc(size);
  int *h_data2_in  = (int *)malloc(size);

  for (int i = 0; i < numElements; ++i) {
    h_data1_in[i] = i;
    h_data2_in[i] = numElements - i;
  }

  int *d_data1_in = NULL;
  err = hipMalloc((void **)&d_data1_in, size);
  
  int *d_data2_in = NULL;
  err = hipMalloc((void **)&d_data2_in, size);

  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_data1_in,  h_data1_in, size, hipMemcpyHostToDevice);
  err = hipMemcpy(d_data2_in,  h_data2_in, size, hipMemcpyHostToDevice);

  int threadsPerBlock = 32;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
  matrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_data1_in,10,2, d_data2_in);
  err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  


  err = hipFree(d_data1_in);
  err = hipFree(d_data2_in);

  free(h_data1_in);
  free(h_data2_in);

  printf("Done\n");
  return 0;
}

